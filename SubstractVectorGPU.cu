#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <iostream>
#include "common.h"

// GPU Kernel to perform vector substraction.
__global__ void VectorSubsKernel(float* ad, float* bd, float* cd, int size)
{
	// Retrieve our coordinates in the block
	int blockId   = blockIdx.y * gridDim.x + blockIdx.x;			 	
	int threadId = blockId * blockDim.x + threadIdx.x; 
	// Perform vector substracion.
	if (threadId<size) 
		cd[threadId] = ad[threadId] - bd[threadId];
}


bool subtractVectorGPU( float* ad, float* bd, float* cd, int size )
{
	// Error return value
	hipError_t status;
	// Number of bytes in the vector.
	int bytes = size * sizeof(float);

	dim3 dimBlock(128, 1);   //Block dimension initialization. 
	
	int gridx = 1;		//x dimension. 
	int gridy = 1;		//y dimension. 
	if(size/128 < 65536)
		gridx = ceil((float)size/128);
	else{
		gridx = 65535;
		gridy = ceil((float)size/(128*65535));
	}
	dim3 dimGrid(gridx, gridy); // Grid initilization. 

	// Launch the kernel on a size-by-size block of threads
	VectorSubsKernel<<<dimGrid, dimBlock>>>(ad, bd, cd, size);
	
	// Wait for completion
	hipDeviceSynchronize();
	// Check for errors
	status = hipGetLastError();
	if (status != hipSuccess) {
		std::cout << "Kernel failed: " << hipGetErrorString(status) << 
		std::endl;
		return false;
	}
	// Success
	return true;
}
